#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <cstdio>

#include "hip/hip_cooperative_groups.h"
#include "hip/hip_runtime_api.h"

#include "motis/core/common/timing.h"

namespace motis {

using namespace cooperative_groups;

using namespace raptor;

// leader type must be unsigned 32bit
// no leader is a zero ballot vote (all 0) minus 1 => with underflow all 1's
constexpr unsigned int FULL_MASK = 0xFFFFffff;
constexpr unsigned int NO_LEADER = FULL_MASK;

__device__ __forceinline__
unsigned int get_block_thread_id() {
  return threadIdx.x + (blockDim.x * threadIdx.y);
}

__device__ __forceinline__
unsigned int get_global_thread_id() {
  return get_block_thread_id() + (blockDim.x * blockDim.y * blockIdx.x);
}

__device__ __forceinline__
unsigned int get_block_stride() {
  return blockDim.x * blockDim.y;
}

__device__ __forceinline__
unsigned int get_global_stride() {
  return get_block_stride() * gridDim.x * gridDim.y;
}

__device__
void mark(unsigned int* store, unsigned int const idx) {
  unsigned int const store_idx = (idx >> 5); // divide by 32
  unsigned int const mask = 1 << (idx % 32); 
  atomicOr(&store[store_idx], mask);
}

__device__
bool marked(unsigned int const * const store, unsigned int idx) {
  unsigned int const store_idx = (idx >> 5); // divide by 32
  unsigned int const val = store[store_idx];
  unsigned int const mask = 1 << (idx % 32);
  return (bool) (val & mask);
}

__device__
void reset_store(unsigned int* store, int const store_size) {
  auto const t_id = get_global_thread_id();
  auto const stride = get_global_stride();
  for (auto idx = t_id; idx < store_size; idx += stride) {
    store[idx] = 0;
  }
}

__device__
void convert_station_to_route_marks(unsigned int* station_marks, 
                                    unsigned int* route_marks) {
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  for (auto idx = global_t_id; idx < GTT.stop_count_; idx += global_stride) {
    if (marked(station_marks, idx)) {
      if (!ANY_STATION_MARKED) { ANY_STATION_MARKED = true; }
      auto const stop =  GTT.stops_[idx];
      for (auto sri = stop.index_to_stop_routes_; 
                sri < stop.index_to_stop_routes_ + stop.route_count_;
              ++sri) {
        mark(route_marks, GTT.stop_routes_[sri]);
      }
    }
  }
}

__device__
motis::time get_stop_arrival(stop_times_index const sti) {
  return GTT.stop_times_[sti].arrival_;
}

__device__
motis::time get_stop_departure(stop_times_index const sti) {
  return GTT.stop_times_[sti].departure_;
}

__device__
motis::time get_stop_arrival_split(stop_times_index const sti) {
  return GTT.stop_arrivals_[sti];
}

__device__
motis::time get_stop_departure_split(stop_times_index const sti) {
  return GTT.stop_departures_[sti];
}

__device__
station_id get_route_stop(route_stops_index const rsi) {
  return GTT.route_stops_[rsi];
}

__device__
station_id get_route_stop_cls(route_stops_index const rsi) {
  return GTT.clustered_route_stops_[rsi];
}

__device__
stop_time get_stop_time(stop_times_index const sti) {
  return GTT.stop_times_[sti];
  // auto const addr = &GTT.stop_times_[idx];
  // uint32_t return_value;

  // asm("ld.global.cg.s32 %0, [%1];" : "=r"(return_value) : "l"(addr));
  // return *((stop_time*) &return_value);
}


__device__ 
motis::time get_arrival(global_mem_time const * const base, 
                        station_id const s_id) {
  // motis::time return_value;
  // asm("ld.global.ca.s16 %0, [%1];" : "=h"(return_value) : "l"(&base[s_id]));
  // return return_value;
  // return __ldg(&base[s_id]);
  return base[s_id];
}

#if SMALL_TIME /* SMALL TIME -- 16 bit time values in global memory */

__device__
bool update_arrival(global_mem_time * const base, 
                    station_id const s_id, 
                    global_mem_time const val) {

#if __CUDA_ARCH__ >= 700

  auto old_value = base[s_id];
  global_mem_time assumed;

  do {
    if (old_value <= val) { return false; }

    assumed = old_value;

    old_value = atomicCAS(&base[s_id], assumed, val);
  } while (assumed != old_value);
  
  return true;

#else

  // we have a 16-bit time value array, but only 32-bit atomic operations
  // therefore every two 16-bit time values are read as one 32-bit time value
  // then they are the corresponding part is updated and stored if a better
  // time value was found while the remaining 16 bit value part remains
  // unchanged

  global_mem_time * const arr_address = &base[s_id];
  unsigned int* base_address = (unsigned int*)((size_t) arr_address & ~2);
  unsigned int old_value, assumed, new_value, compare_val;

  old_value = *base_address;

  do {
    assumed = old_value;

    if ((size_t) arr_address & 2) {
      compare_val = (0x0000FFFF & assumed) ^ (((unsigned int) val) << 16);
    } else {
      compare_val = (0xFFFF0000 & assumed) ^ (unsigned int) val;
    }

    new_value = __vminu2(old_value, compare_val);

    if (new_value == old_value) { return false; }

    old_value = atomicCAS(base_address, assumed, new_value);
  } while (assumed != old_value);

  return true;

#endif
}

#else /* BIG TIME -- 32bit time values in global memory */

__device__
motis::time get_arrival(global_mem_time const * const base, 
                        station_id const s_id) {
  return (motis::time) (base[s_id]);
}

__device__
void update_arrival(global_mem_time* const base, 
                    station_id const s_id, 
                    motis::time const val) {
  global_mem_time* address = &base[s_id];
  atomicMin(address, (motis::time32) val);
}

#endif /* END IF STATEMENT */

__device__
void reset_arrivals(motis::time* const arrivals) {
  auto const global_stride = get_global_stride();
  auto arr_idx = get_global_thread_id();

  for (; arr_idx < GTT.stop_count_; arr_idx += global_stride) {
    arrivals[arr_idx] = invalid<global_mem_time>;
  }
}

__device__
void copy_marked_arrivals(motis::time* const to, motis::time* const from, unsigned int* station_marks) {
  auto const global_stride = get_global_stride();

  auto arr_idx = get_global_thread_id();
  for (; arr_idx < GTT.stop_count_; arr_idx += global_stride) {
    if (marked(station_marks, arr_idx)) {
      to[arr_idx] = from[arr_idx]; 
    } else {
      to[arr_idx] = invalid<motis::time>;
    }
  }
}

__device__
void copy_arrivals(motis::time* const to, motis::time* const from) {
  auto const global_stride = get_global_stride();

  auto arr_idx = get_global_thread_id();
  for (; arr_idx < GTT.stop_count_; arr_idx += global_stride) {
    to[arr_idx] = from[arr_idx]; 
  }
}

__device__
void copy_and_min_arrivals(motis::time* const to, motis::time* const from) {
  auto const global_stride = get_global_stride();

  auto arr_idx = get_global_thread_id();
  for (; arr_idx < GTT.stop_count_; arr_idx += global_stride) {
    to[arr_idx] = min(from[arr_idx], to[arr_idx]); 
  }
}

template<GetArrivalFun GetArrival, 
         UpdateArrivalFun UpdateArrival,
         GetRouteStopFun GetRouteStop,
         GetStopArrivalFun GetStopArrival, 
         GetStopDepartureFun GetStopDeparture>
__device__
void update_route_larger32_gen(gpu_route const& route,
                               global_mem_time const * const prev_arrivals,
                               global_mem_time* const arrivals,
                               unsigned int* station_marks) {
  auto const t_id = threadIdx.x;

  station_id station_id_t = invalid<station_id>;
  motis::time prev_arrival = invalid<motis::time>;
  motis::time stop_arrival = invalid<motis::time>;
  motis::time stop_departure = invalid<motis::time>;
  
  int active_stop_count = route.stop_count_;

  // this is ceil(stop_count / 32)
  int const stage_count = (route.stop_count_ + (32 - 1)) >> 5;
  int active_stage_count = stage_count;

  unsigned int leader = NO_LEADER;
  unsigned int any_arrival = 0;

  for (int trip_offset = 0; 
               trip_offset < route.trip_count_; 
             ++trip_offset) {

    for (int current_stage = 0; 
             current_stage < active_stage_count; 
           ++current_stage) {

      int stage_id = (current_stage << 5) + t_id;

      // load the prev arrivals for the current stage
      if (stage_id < active_stop_count) {
        station_id_t = GetRouteStop(route.index_to_route_stops_ + stage_id);
        prev_arrival = GetArrival(prev_arrivals, station_id_t);
      } 

      any_arrival |= __any_sync(FULL_MASK, valid(prev_arrival));
      if (current_stage == active_stage_count - 1 && !any_arrival) { return; }
      if (!any_arrival) { continue; }

      // load the stop times for the current stage
      if (stage_id < active_stop_count) {
        auto const st_idx = route.index_to_stop_times_
                    + (trip_offset * route.stop_count_)
                    + stage_id;
        stop_departure = GetStopDeparture(st_idx);
      } 

      // get the current stage leader
      unsigned int ballot = __ballot_sync(FULL_MASK,
                                    (stage_id < active_stop_count) 
                                  && valid(prev_arrival) 
                                  && valid(stop_departure)
                                  && (prev_arrival <= stop_departure));
      leader = __ffs(ballot) - 1;

      if (leader != NO_LEADER) {
        leader += current_stage << 5;
      } 

      // first update the current stage
      if (leader != NO_LEADER && stage_id < active_stop_count) {

        if (stage_id > leader) {
        auto const st_idx = route.index_to_stop_times_
                    + (trip_offset * route.stop_count_)
                    + stage_id;
          stop_arrival = GetStopArrival(st_idx);
          bool updated = UpdateArrival(arrivals, station_id_t, stop_arrival);
          if (updated) { mark(station_marks, station_id_t); }
        } 
      }

      // then update all upward stages
      if (leader != NO_LEADER) {
        for (int upward_stage = current_stage + 1; 
                 upward_stage < active_stage_count;
               ++upward_stage) {

          int upwards_id = (upward_stage << 5) + t_id;
          if (upwards_id < active_stop_count) {

          auto const st_idx = route.index_to_stop_times_
                          + (trip_offset * route.stop_count_) 
                          + upwards_id;

            stop_arrival = GetStopArrival(st_idx);
            station_id_t = GetRouteStop(route.index_to_route_stops_ 
                                        + upwards_id);
            bool updated = UpdateArrival(arrivals, station_id_t, stop_arrival);
            if (updated) { mark(station_marks, station_id_t); }
          } 
        }

        // for this route we do not need to update any station higher than the leader anymore
        active_stop_count = leader;
        active_stage_count = (active_stop_count + (32 - 1)) >> 5;
        leader = NO_LEADER;
      } 
    }
  }
}

template<GetArrivalFun GetArrival, 
         UpdateArrivalFun UpdateArrival,
         GetRouteStopFun GetRouteStop, 
         GetStopArrivalFun GetStopArrival,
         GetStopDepartureFun GetStopDeparture>
__device__
void update_route_smaller32_fronts_gen(gpu_route const route,
                                   global_mem_time const * const prev_arrivals,
                                   global_mem_time * const arrivals,
                                   unsigned int* station_marks) {
  auto const t_id = threadIdx.x;
  auto const index_to_stop_times = route.index_to_stop_times_;
  auto const stop_count = route.stop_count_;

  station_id station_id_t = invalid<station_id>;
  motis::time prev_arrival = invalid<motis::time>;
  motis::time stop_arrival = invalid<motis::time>;
  motis::time stop_departure = invalid<motis::time>;

  unsigned leader = stop_count;
  unsigned int active_stop_count = stop_count;

  unsigned const front_count = 32 / stop_count;
  unsigned const front_lane = threadIdx.x % stop_count;
  unsigned const excess_threshold = front_count * stop_count;

  unsigned const front_id = threadIdx.x / stop_count;
  stop_times_index st_idx = invalid<stop_times_index>;

  if (front_lane < active_stop_count && t_id < excess_threshold) {
    station_id_t = GTT.route_stops_[route.index_to_route_stops_ + front_lane];
    prev_arrival = GetArrival(prev_arrivals, station_id_t);
  }

  if (!__any_sync(FULL_MASK, valid(prev_arrival))) { return; }

  for (unsigned trip_offset = front_id; 
                trip_offset < route.trip_count_;
                trip_offset += front_count) {

    if (front_lane < active_stop_count && t_id < excess_threshold) {
      st_idx = index_to_stop_times 
             + (trip_offset * stop_count) 
             + front_lane;
      stop_departure = GetStopDeparture(st_idx);
    }

    unsigned ballot = __ballot_sync(FULL_MASK, 
                                    (front_lane < active_stop_count)
                                 && (t_id < excess_threshold)
                                 && valid(prev_arrival)
                                 && valid(stop_departure)
                                 && (prev_arrival <= stop_departure));

    for (unsigned f_id = 0; f_id < front_count; ++f_id) {
      if (f_id == front_id) {

        leader = __ffs(ballot) - 1;

        if (front_lane > leader && front_lane < active_stop_count) {
          stop_arrival = GetStopArrival(st_idx);
          bool updated = UpdateArrival(arrivals, station_id_t, stop_arrival);
          if (updated) { mark(station_marks, station_id_t); }
        } 

        if (leader != NO_LEADER) {
          active_stop_count = leader;
        }
      }
      
      ballot = ballot >> stop_count;
    }
  }
}

template<GetArrivalFun GetArrival, 
         UpdateArrivalFun UpdateArrival,
         GetRouteStopFun GetRouteStop, 
         GetStopArrivalFun GetStopArrival,
         GetStopDepartureFun GetStopDeparture>
__device__
void update_route_smaller32_gen(gpu_route const route,
                                global_mem_time const * const prev_arrivals,
                                global_mem_time * const arrivals,
                                unsigned int* station_marks) {
  auto const t_id = threadIdx.x;

  station_id station_id_t = invalid<station_id>;
  motis::time prev_arrival = invalid<motis::time>;
  motis::time stop_arrival = invalid<motis::time>;
  motis::time stop_departure = invalid<motis::time>;

  unsigned leader = route.stop_count_;
  unsigned int active_stop_count = route.stop_count_;

  if (t_id < active_stop_count) {
    station_id_t = GetRouteStop(route.index_to_route_stops_ + t_id);
    prev_arrival = GetArrival(prev_arrivals, station_id_t);
  }

  if (!__any_sync(FULL_MASK, valid(prev_arrival))) { return; }

  for (int trip_offset = 0; 
           trip_offset < route.trip_count_; 
         ++trip_offset) {

    if (t_id < active_stop_count) {
      auto const st_idx = route.index_to_stop_times_ 
                        + (trip_offset * route.stop_count_) 
                        + t_id;
      stop_departure = GetStopDeparture(st_idx);
    }

    // elect leader
    unsigned ballot = __ballot_sync(FULL_MASK, 
                                    (t_id < active_stop_count)
                                 && valid(prev_arrival)
                                 && valid(stop_departure)
                                 && (prev_arrival <= stop_departure));
    leader = __ffs(ballot) - 1;

    if (t_id > leader && t_id < active_stop_count) {
      auto const st_idx = route.index_to_stop_times_ 
                        + (trip_offset * route.stop_count_) 
                        + t_id;

      stop_arrival = GetStopArrival(st_idx);
      bool updated = UpdateArrival(arrivals, station_id_t, stop_arrival);
      if (updated) { mark(station_marks, station_id_t); }
    } 

    if (leader != NO_LEADER) {
      active_stop_count = leader;
    }
    leader = NO_LEADER;
  }
}

template <GetArrivalFun GetArrival, UpdateArrivalFun UpdateArrival>
__device__ void update_footpaths_dev_scratch_gen(
    global_mem_time const* const read_arrivals,
    global_mem_time* const write_arrivals, unsigned int* station_marks) {

  auto const global_stride = get_global_stride();

  auto foot_idx = get_global_thread_id();
  for (; foot_idx < GTT.footpath_count_; foot_idx += global_stride) {
    auto const footpath = GTT.footpaths_[foot_idx];

    motis::time const from_arrival = GetArrival(read_arrivals, footpath.from_);
    motis::time const new_arrival = from_arrival + footpath.duration_;

    if (valid(from_arrival) && marked(station_marks, footpath.from_)) {
      bool updated = UpdateArrival(write_arrivals, footpath.to_, new_arrival);
      if (updated) { mark(station_marks, footpath.to_); }
    }
  }
}

__device__
auto const update_footpaths_dev_scratch
        =  update_footpaths_dev_scratch_gen<get_arrival, update_arrival>;

template<GetArrivalFun GetArrival, UpdateArrivalFun UpdateArrival>
__device__
void update_footpaths_dev_new_gen(global_mem_time const * const arrivals,
                                  global_mem_time* const next_arrivals,
                                  unsigned int* station_marks) {
  auto const global_stride = get_global_stride();

  auto foot_idx = get_global_thread_id();
  for (; foot_idx < GTT.footpath_count_; foot_idx += global_stride) {
    auto const footpath = GTT.footpaths_[foot_idx];

    motis::time from_arrival = arrivals[footpath.from_];
    motis::time new_arrival = from_arrival + footpath.duration_;

    if (valid(from_arrival) && marked(station_marks, footpath.from_)) {
      bool updated = UpdateArrival(next_arrivals, footpath.to_, new_arrival);
      if (updated) { mark(station_marks, footpath.to_); }
    }
  }
}

__device__
auto const update_footpaths_dev_new
        =  update_footpaths_dev_new_gen<get_arrival, update_arrival>;

__device__
auto const update_route_smaller32
         = update_route_smaller32_gen<get_arrival, 
                                      update_arrival,  
                                      get_route_stop, 
                                      get_stop_arrival, 
                                      get_stop_departure>;
__device__
auto const update_route_larger32
         = update_route_larger32_gen<get_arrival, 
                                      update_arrival,
                                      get_route_stop, 
                                      get_stop_arrival,
                                      get_stop_departure>;

__device__
auto const update_route_smaller32_cls
         = update_route_smaller32_gen<get_arrival, 
                                      update_arrival,
                                      get_route_stop_cls, 
                                      get_stop_arrival,
                                      get_stop_departure>;
__device__
auto const update_route_larger32_cls
         = update_route_larger32_gen<get_arrival, 
                                     update_arrival,
                                     get_route_stop_cls,
                                     get_stop_arrival,
                                     get_stop_departure>;
__device__
auto const update_route_smaller32_cls_split
         = update_route_smaller32_gen<get_arrival, 
                                      update_arrival,
                                      get_route_stop_cls, 
                                      get_stop_arrival_split,
                                      get_stop_departure_split>;
__device__
auto const update_route_larger32_cls_split
         = update_route_larger32_gen<get_arrival, 
                                     update_arrival,
                                     get_route_stop_cls,
                                     get_stop_arrival_split,
                                     get_stop_departure_split>;

__device__
auto const update_route_smaller32_fronts
         = update_route_smaller32_fronts_gen<get_arrival, 
                                             update_arrival,
                                             get_route_stop,
                                             get_stop_arrival,
                                             get_stop_departure>;

__device__
auto const update_route_smaller32_split
         = update_route_smaller32_gen<get_arrival, 
                                      update_arrival,  
                                      get_route_stop, 
                                      get_stop_arrival_split, 
                                      get_stop_departure_split>;

__device__
auto const update_route_larger32_split
         = update_route_larger32_gen<get_arrival, 
                                      update_arrival,
                                      get_route_stop, 
                                      get_stop_arrival_split,
                                      get_stop_departure_split>;

__device__
void update_routes_dev(global_mem_time const * const prev_arrivals,
                       global_mem_time* const arrivals,
                       unsigned int* station_marks,
                       unsigned int* route_marks) {

  if (get_global_thread_id() == 0) { ANY_STATION_MARKED = false; }

  convert_station_to_route_marks(station_marks, route_marks);
  this_grid().sync();

  auto const station_store_size = (GTT.stop_count_ / 32) + 1;
  reset_store(station_marks, station_store_size);
  this_grid().sync();

  if (!ANY_STATION_MARKED) { return; }

  auto const stride = blockDim.y * gridDim.x;
  auto const start_r_id = threadIdx.y + (blockDim.y * blockIdx.x);
  for (auto r_id = start_r_id; r_id < GTT.route_count_; r_id += stride) {
    if (!marked(route_marks, r_id)) { continue; }

    auto const route = GTT.routes_[r_id];
    if (route.stop_count_ <= 32) {
      // update_route_smaller32_split(route, prev_arrivals, arrivals, r_id);
      update_route_smaller32_split(route, 
                                   prev_arrivals, arrivals,
                                   station_marks);
    } else {
      // update_route_larger32_split(route, prev_arrivals, arrivals, r_id);
      update_route_larger32_split(route, 
                                  prev_arrivals, arrivals,
                                  station_marks);
    }
  }  

  this_grid().sync();

  auto const store_size = (GTT.route_count_ / 32) + 1;
  reset_store(route_marks, store_size);
}

__device__
void init_arrivals_dev(d_query const& dq) {
  auto const t_id = get_global_thread_id();

  auto const station_store_size = (GTT.stop_count_ / 32) + 1;
  reset_store(dq.station_marks_, station_store_size);

  auto const route_store_size = (GTT.route_count_ / 32) + 1;
  reset_store(dq.route_marks_, route_store_size);

  if (t_id == 0) { ANY_STATION_MARKED = false; }

  if (t_id == 0) {
    dq.d_arrivals_[0][dq.source_] = dq.source_time_begin_;
    mark(dq.station_marks_, dq.source_);
  }

  auto const footpath_count =
      GTT.initialization_footpaths_indices_[dq.source_ + 1] -
      GTT.initialization_footpaths_indices_[dq.source_];
  if (t_id < footpath_count) {
    auto const index_into_footpaths =
               GTT.initialization_footpaths_indices_[dq.source_];
    auto const f = GTT.initialization_footpaths_[index_into_footpaths + t_id];

    global_mem_time const new_value = dq.source_time_begin_ + f.duration_;
    bool updated = update_arrival(dq.d_arrivals_[0], f.to_, new_value);
    if (updated) { mark(dq.station_marks_, f.to_); }
  }
}

__device__
void update_footpaths_dev(d_query const dq, int round_k) {
  global_mem_time* const arrivals = dq.d_arrivals_[round_k];
  global_mem_time* const next_arrivals = dq.d_arrivals_[round_k + 1];

  // we must only copy the marked arrivals, 
  // since an earlier raptor query might have used a footpath
  // to generate the current arrival, a new optimum from this value 
  // would be generated using a double walk -> not correct!
  copy_marked_arrivals(dq.footpaths_scratchpad_, arrivals, dq.station_marks_);
  this_grid().sync();

  update_footpaths_dev_scratch(dq.footpaths_scratchpad_, arrivals,
                               dq.station_marks_);
  this_grid().sync();

  copy_and_min_arrivals(next_arrivals, arrivals);
  this_grid().sync();
}

__global__
__launch_bounds__((block_dim_x * block_dim_y), min_blocks_per_sm)
void init_arrivals_kernel(d_query const d_query) {
  init_arrivals_dev(d_query);
}

__global__
__launch_bounds__((block_dim_x * block_dim_y), min_blocks_per_sm)
void update_footpaths_kernel(d_query const dq, int round_k) {
  update_footpaths_dev(dq, round_k);
}

__global__
__launch_bounds__((block_dim_x * block_dim_y), min_blocks_per_sm)
void update_routes_kernel(d_query const dq, int round_k) {
  global_mem_time const * const prev_arrivals = dq.d_arrivals_[round_k - 1];
  global_mem_time* const arrivals = dq.d_arrivals_[round_k];

  update_routes_dev(prev_arrivals, arrivals, 
                    dq.station_marks_, dq.route_marks_);
}

__global__
__launch_bounds__((block_dim_x * block_dim_y), min_blocks_per_sm)
void gpu_raptor_kernel(d_query const dq) {
  init_arrivals_dev(dq);
  this_grid().sync();

  for (int8_t round_k = 1; round_k < max_round_k; ++round_k) {
    const global_mem_time* const prev_arrivals = dq.d_arrivals_[round_k-1];
    global_mem_time* const arrivals = dq.d_arrivals_[round_k];
    global_mem_time* const next_arrivals = dq.d_arrivals_[round_k + 1];

    update_routes_dev(prev_arrivals, arrivals,
                      dq.station_marks_, dq.route_marks_);
    this_grid().sync();

    update_footpaths_dev(dq, round_k);
  }
}

void invoke_gpu_raptor(d_query& dq) {
  void* kernel_args[] = {(void*)&dq};
  hipLaunchCooperativeKernel((void*)gpu_raptor_kernel, grid_dim,
                              threads_per_block, kernel_args);
  cc();
  hipDeviceSynchronize();
  cc();

  fetch_result_from_device(dq);
}

} // namespace motis